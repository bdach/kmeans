#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include "stdio.h"

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <vector>

inline void on_error(hipError_t errcode, const char *file, int line) {
	if (errcode != hipSuccess) {
		fprintf(stderr, "CUDA error: %s (%s:%d)\n", hipGetErrorString(errcode), file, line);
		exit(EXIT_FAILURE);
	}
}

#define checkCudaErrors(ret) on_error((ret), __FILE__, __LINE__)
#define getLastCudaError() on_error(hipGetLastError(), __FILE__, __LINE__)
#define NUM_THREADS 256

extern "C" void run_kernel(unsigned int n,
		unsigned int k,
		float tolerance,
		const float *in_x,
		const float *in_y,
		const float *in_z,
		float *out_x,
		float *out_y,
		float *out_z);

__global__ void calculate_distances(unsigned int n,
		unsigned int k,
		float **points,
		float **means,
		unsigned int *membership,
		unsigned char *subdelta)
{
	extern __shared__ unsigned char membership_changed[];

	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tid = threadIdx.x;
	if (idx >= n) return;
	float min_dist = INFINITY;
	unsigned int cluster = 0;
	for (unsigned int i = 0; i < k; ++i) {
		float dx = points[0][idx] - means[0][i];
		float dy = points[1][idx] - means[1][i];
		float dz = points[2][idx] - means[2][i];
		float dist = dx * dx + dy * dy + dz * dz;
		if (dist < min_dist) {
			min_dist = dist;
			cluster = i;
		}
		__syncthreads(); // end of uncertain branch
	}
	membership_changed[tid] = membership[idx] == cluster;
	membership[idx] = cluster;
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s)
			membership_changed[tid] += membership_changed[tid + s];
		__syncthreads();
	}
	if (tid == 0) {
		subdelta[blockIdx.x] = membership_changed[0];
	}
}

extern "C" void run_kernel(unsigned int n,
		unsigned int k,
		float tolerance,
		const float *in_x,
		const float *in_y,
		const float *in_z,
		float *out_x,
		float *out_y,
		float *out_z)
{
	const unsigned int points_size = n * sizeof(float);
	const float *points[3] = {in_x, in_y, in_z};
	float *points_d[3];
	float **d_points_d;
	for (int i = 0; i < 3; ++i) {
		checkCudaErrors(hipMalloc((void **)&points_d[i], points_size));
		checkCudaErrors(hipMemcpy(points_d[i], points[i], points_size, hipMemcpyHostToDevice));
	}
	checkCudaErrors(hipMalloc((void **)&d_points_d, sizeof(points_d)));
	checkCudaErrors(hipMemcpy(d_points_d, points_d, sizeof(points_d), hipMemcpyHostToDevice));

	const unsigned int means_size = k * sizeof(float);
	float *means[3] = {out_x, out_y, out_z};
	float *means_d[3];
	float **d_means_d;
	for (unsigned int i = 0; i < 3; ++i) {
		checkCudaErrors(hipMalloc((void **)&means_d[i], means_size));
		checkCudaErrors(hipMemcpy(means_d[i], means[i], means_size, hipMemcpyHostToDevice));
	}
	checkCudaErrors(hipMalloc((void **)&d_means_d, sizeof(means_d)));
	checkCudaErrors(hipMemcpy(d_means_d, means_d, sizeof(means_d), hipMemcpyHostToDevice));

	unsigned int block_count = ceil((float)n / NUM_THREADS);
	unsigned int subdelta_size = block_count * sizeof(unsigned char);
	unsigned int shared_subdelta_size = NUM_THREADS * sizeof(unsigned char);
	unsigned char *d_subdelta; // WARNING: This works because NUM_THREADS is 256
	checkCudaErrors(hipMalloc((void **)&d_subdelta, subdelta_size));

	const unsigned int membership_size = n * sizeof(unsigned int);
	unsigned int *new_membership, *d_new_membership;
	new_membership = (unsigned int *)malloc(membership_size);
	checkCudaErrors(hipMalloc((void **)&d_new_membership, membership_size));

	unsigned int delta = n;
	while (((float)delta / n) > tolerance) {
		delta = 0;
		calculate_distances<<< block_count, NUM_THREADS, shared_subdelta_size >>>(n, k, d_points_d, d_means_d, d_new_membership, d_subdelta);
		getLastCudaError();
		thrust::device_ptr<unsigned char> ptr(d_subdelta);
		delta = thrust::reduce(ptr, ptr + block_count);
		std::vector<unsigned int> counts(k);
		for (unsigned int j = 0; j < k; ++j) {
			means[0][j] = 0;
			means[1][j] = 0;
			means[2][j] = 0;
		}
		checkCudaErrors(hipMemcpy(new_membership, d_new_membership, membership_size, hipMemcpyDeviceToHost));
		for (unsigned int i = 0; i < n; ++i) {
			unsigned int cluster = new_membership[i];
			means[0][cluster] += points[0][i];
			means[1][cluster] += points[1][i];
			means[2][cluster] += points[2][i];
			counts[cluster] += 1;
		}
		for (unsigned int j = 0; j < k; ++j) {
			means[0][j] /= counts[j];
			means[1][j] /= counts[j];
			means[2][j] /= counts[j];
		}
		for (unsigned int i = 0; i < 3; ++i) {
			checkCudaErrors(hipMemcpy(means_d[i], means[i], means_size, hipMemcpyHostToDevice));
		}
	}

	for (unsigned int i = 0; i < 3; ++i) {
		checkCudaErrors(hipFree(points_d[i]));
		checkCudaErrors(hipFree(means_d[i]));
	}
	checkCudaErrors(hipFree(d_points_d));
	checkCudaErrors(hipFree(d_means_d));
	checkCudaErrors(hipFree(d_new_membership));
	checkCudaErrors(hipFree(d_subdelta));
	free(new_membership);
}
